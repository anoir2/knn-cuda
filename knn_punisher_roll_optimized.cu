
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include <hip/hip_runtime_api.h>

#define restrict __restrict__
#define PADDINGCLASS -2
#define OUTPUT_FILE "ocuda"
#define INPUT_FILE "data"
#define KMAX 20
#define CLASSESMAX 100
#define SPACEDIMMAX 100

void check_error(hipError_t err, const char *msg);
void printStats(hipEvent_t before, hipEvent_t after, const char *msg);
void readInput(FILE* file, float* coords, int* classes, int spacedim, int numels, int totalElements);
void writeOutput(float* coords, int* classes, int spacedim, int numels);

__global__ void knn(float* const restrict coords, float2* restrict kOutput, const int totalElements, const int numels, const int spacedim, const int k, int* restrict classes, const int classes_num);
__global__ void knnPunisher(float2* restrict kOutput, int* restrict classes, const int numels, const int newels, const int k, const int classes_num);
__device__ float distance(float* const coords, float* const coords2, const int spacedim);
__device__ int insert(float* kPoints, float2 newDist, int* size, const int k, const int gid, const int offset);
__device__ void swap(float* x1, float* x2, float* y1, float* y2);
__device__ int deviceFindMode(int* kclasses, int classes_num, int k);
__device__ float distanceShm(float* coords, int left, int spacedim);

int main(int argc, char *argv[])
{  
  int newels;                      //number of points we want classify
  int k;                           //number of nearest points we use to classify
  int numels;                      //total element already classified
  int spacedim;
  char filePath[255];              //path + filname of input file
  int classes_num;                 //number of classes
  float* h_coords;                //coords of existing points with a class
  int* h_classes;                  //array contains the class for each points
  
  //*** Device-variables-declaration ***
  float* d_coords;
  int2* d_determinate;
  int* d_classes;
  float2* d_kOutput;
  //*** end-device-declaration
  
  //***cudaEvent-declaration***
  hipEvent_t before_allocation, before_input, before_upload, before_knn, before_download;
  hipEvent_t after_allocation, after_input, after_upload, after_knn, after_download;
  //***end-cudaEvent-declaration***
  
  //Requisiti: numels e newels devono essere maggiori di K
  if (argc > 2) 
  {
    strcpy(filePath, argv[1]);
    k = atoi(argv[2]);
  }
  else 
  {
    printf("how-to-use: knn <inputfile> <k> \n");
    exit(1);
  } 
  
  //***cuda-init-event***
  check_error(hipEventCreate(&before_allocation), "create before_allocation cudaEvent");
  check_error(hipEventCreate(&before_input), "create before_input cudaEvent");
  check_error(hipEventCreate(&before_upload), "create before_upload cudaEvent");
  check_error(hipEventCreate(&before_knn), "create before_knn cudaEvent");
  check_error(hipEventCreate(&before_download), "create before_download cudaEvent");
  
  check_error(hipEventCreate(&after_allocation), "create after_allocation cudaEvent");
  check_error(hipEventCreate(&after_input), "create after_input cudaEvent");
  check_error(hipEventCreate(&after_upload), "create after_upload cudaEvent");
  check_error(hipEventCreate(&after_knn), "create after_knn cudaEvent");
  check_error(hipEventCreate(&after_download), "create after_download cudaEvent");
  //***end-cuda-init-event***

  FILE *fp;
  if((fp = fopen(filePath, "r")) == NULL)
  {
        printf("No such file\n");
        exit(1);
  }
  
  fseek(fp, 0L, SEEK_END);
  float fileSize = ftell(fp);
  rewind(fp);
  
  int count = fscanf(fp, "%d,%d,%d,%d\n", &numels, &newels, &classes_num, &spacedim);
  int totalElements = numels + newels;

  //*** allocation ***
  hipEventRecord(before_allocation);
  h_coords = (float*) malloc(sizeof(float)*totalElements*spacedim);
  h_classes = (int*) malloc(sizeof(int)*totalElements);
   
  //*** device-allocation ***
  check_error(hipMalloc(&d_coords, totalElements*spacedim*sizeof(float)), "alloc d_coords_x");
  check_error(hipMalloc(&d_classes, totalElements*sizeof(int)), "alloc d_classes");
  check_error(hipMalloc(&d_determinate, newels*2*sizeof(int)), "alloc d_determinate");
  check_error(hipMalloc(&d_kOutput, newels*KMAX*2*sizeof(float)), "alloc d_kOutput");

  //*** end-device-allocation ***
  hipEventRecord(after_allocation);
  
  ///***input-from-file***
  hipEventRecord(before_input);
  readInput(fp, h_coords, h_classes, spacedim, numels, totalElements);
  hipEventRecord(after_input);
  fclose(fp);
  ///***end-input-from-file***

  //***copy-arrays-on-device***
  hipEventRecord(before_upload);
  check_error(hipMemcpy(d_coords, h_coords, totalElements*spacedim*sizeof(float), hipMemcpyHostToDevice), "copy d_coords");
  check_error(hipMemcpy(d_classes, h_classes, totalElements*sizeof(int), hipMemcpyHostToDevice), "copy d_classes");
  hipEventRecord(after_upload);
  //***end-copy-arrays-on-device***                              
  
  int blockSize = 512;
  int numBlocks = (newels + blockSize - 1)/blockSize;
   
  hipEventRecord(before_knn);
  knn<<<numBlocks, blockSize, (blockSize*sizeof(float)*2*k)>>>(d_coords, d_kOutput, totalElements, numels, spacedim, k, d_classes, classes_num);
  blockSize = 32;
  numBlocks = (newels + blockSize - 1)/blockSize;
  knnPunisher<<<numBlocks, blockSize, newels*sizeof(int)>>>(d_kOutput, d_classes, numels, newels, k, classes_num);
  hipEventRecord(after_knn);
  check_error(hipMemcpy(h_classes+numels, d_classes+numels, newels*sizeof(int), hipMemcpyDeviceToHost), "download classes");

  check_error(hipEventSynchronize(after_knn), "sync cudaEvents");
  printStats(before_knn, after_knn, "knn");
    
  writeOutput(h_coords, h_classes, spacedim, totalElements);
  return 0;
}

void check_error(hipError_t err, const char *msg)
{
  if (err != hipSuccess) 
  {
    fprintf(stderr, "%s : error %d (%s)\n", msg, err, hipGetErrorString(err));
    exit(err);
  }
}

float runtime;
void printStats(hipEvent_t before, hipEvent_t after, const char *msg)
{ 
  check_error(hipEventElapsedTime(&runtime, before, after), msg);
  printf("%s %gms\n", msg, runtime);
}

__global__ void knn(float* const restrict coords, float2* restrict kOutput, const int totalElements, const int numels, const int spacedim, const int k, int* restrict classes, const int classes_num)
{
  extern __shared__ float shm[];
  int gid = numels + threadIdx.x + blockIdx.x*blockDim.x;
  if (gid >= totalElements) return;
  
  float* newPointCoords = coords+spacedim*gid;
  float* pointCoords;
  int offset = blockDim.x;
  int lid = threadIdx.x;

  int i = 0, size = 0, count = 0;
  float point[SPACEDIMMAX];

  for (i = 0; i < spacedim; i++)
   point[i] = newPointCoords[i];
  
  float2 dist;
  pointCoords = coords;
  for (i = 0; i < numels; i++)
  {
    dist = make_float2(distance(point, pointCoords, spacedim), i);
    insert(shm+lid, dist, &size, k, gid, offset);
    pointCoords += spacedim;
  }
  
  for (count=0; i < gid; i++)
  {
      dist = make_float2(distance(point, pointCoords, spacedim), i);
      count += insert(shm+lid, dist, &size, k, gid, offset);
      pointCoords += spacedim;
  }
    
  if (count > 0)
  {
    classes[gid] = -1;
  }
  else
  {
    int kclasses[KMAX];
    for (int j = 0; j < k; j++)
      kclasses[j] = classes[(int)(shm[(j+k)*offset+lid])];
    classes[gid] = deviceFindMode(kclasses, classes_num, k);
  }
  
  //copia kPoints in kOutput
  int newelId = gid-numels;
  for (i = 0; i < k; i++)
  {
    kOutput[newelId*KMAX + i].x = shm[i*offset + lid];
    kOutput[newelId*KMAX + i].y = shm[(i+k)*offset + lid];
  }
    
}


__global__ void knnPunisher(float2* restrict kOutput, int* restrict classes, const int numels, const int newels, const int k, const int classes_num)
{
  //Declaration of shared-memory. It's going to contains partial minimum of distances
  extern __shared__ int mPartial[];
  
  int gid = threadIdx.x + blockIdx.x*blockDim.x;
  if (gid >= newels) return;

  //stiamo lancio un kernel per ogni newels
  //prendiamo la classe del newels e la mettiamo nella shared memory
  int i, id, lid, kclasses[KMAX], kPoints[KMAX], count = 1;
  
  //Se sono qui la classe per il kPoint è da determinare
  for (i = 0; i < k; i++)
    kPoints[i] = kOutput[gid*KMAX+i].y; //gid

  while(count != 0)
  {
      for (i = 0; i < gid; i++)
              mPartial[i] = classes[i+numels];

      mPartial[gid] = classes[gid+numels];
      if (mPartial[gid] != -1) return;
      __syncthreads();

      //Le sue dipendenze, se già determinate stanno nella shared-memory
      count = 0;
      for (i = k-1; i >= 0; i--)
      {
        id = kPoints[i];
        lid = id - numels;
        if (id > numels && mPartial[lid] < 0)
        {
          //segno quelli indeterminati
          count++;
          break;
        }
      }

      if (count == 0)
      {
        //posso determinare il punto
        //le sue dipendenze si trovano in shared memory
          for (i = 0; i < k; i++)
              kclasses[i] = classes[kPoints[i]];
        classes[gid+numels] = deviceFindMode(kclasses, classes_num, k);
      }
  }
}

__device__ int deviceFindMode(int* kclasses, int classes_num, int k)
{
  int classesCount[CLASSESMAX];

  int i;
  int temp=0;
  
  for (i = 0; i < CLASSESMAX; i++)
    classesCount[i] = 0;
       
  for (i = 0; i < k; i++){
    temp=kclasses[i];
    classesCount[temp]+=1;
  } 

  int max = 0;
  int maxValue = classesCount[0];

  for (i = 1; i < classes_num; i++)
  {
    int value = classesCount[i];
    if (value > maxValue)
    {
      max = i;
      maxValue = value;
    }
    else if (value != 0 && maxValue == value)
    {
        int j = 0;
        for (j = 0; j < k; j++)
        {
          if (kclasses[j] == i)
          {
            max = i;
            break;
          }
          else if (kclasses[j] == max)
            break;
        }
    }
  }
  
  return max;
}

//inserimento smart in kPoints
__device__ int insert(float* kPoints, float2 newDist, int* size, const int k, const int gid, const int offset)
{  
    int inserted = 0;
    if (*size == 0)
    {
      //Caso base: inserimento su array vuoto
      kPoints[0] = newDist.x;
      kPoints[k*offset] = newDist.y;
      *size = *size + 1;
      return 1;
    }
  
    int i = 1;
    float* valueX, *valueY, *tailX, *tailY;
    valueX = &(newDist.x);
    valueY = &(newDist.y);
    tailX = &(kPoints[(*size-i)*offset]);
    tailY = &(kPoints[(*size-i+k)*offset]);

    if (*size < k)
    {  
        kPoints[(*size)*offset] = newDist.x;
        kPoints[((*size)+k)*offset] = newDist.y;
        valueX = &(kPoints[(*size)*offset]);
        valueY = &(kPoints[((*size)+k)*offset]);
        inserted = 1;
    }

    //partire della fine, swap se trovo elemento più grande - mi fermo se trovo elemento più piccolo
    while (i <= *size && *(tailX) > *(valueX))
    {
        swap(tailX, valueX, tailY, valueY);
        valueX = tailX;
        valueY = tailY;
        i++;
        tailX = &(kPoints[(*size-i)*offset]);
        tailY = &(kPoints[(*size-i+k)*offset]);
        inserted = 1;    
    }
    
    if (inserted && *size < k) *size = *size + 1;
    return inserted;
}

__device__ void swap(float* x1, float* x2, float* y1, float* y2)
{
  float tmp;
  
  tmp = *(x1);
  *(x1) = *(x2);
  *(x2) = tmp;
  
  tmp = *(y1);
  *(y1) = *(y2);
  *(y2) = tmp;
}


// read input from file
void readInput(FILE* file, float* coords, int* classes, int spacedim, int numels, int totalElements)
{
  int i, j;
  int count;
  for(i=0; i<numels; i++)
  {
    for (j = 0; j < spacedim; j++)
      count = fscanf(file, "%f,", &(coords[i*spacedim +j]));
    count = fscanf(file, "%d\n", &(classes[i]));
  }
   
  for(; i < totalElements; i++)
  {
    for (j = 0; j < spacedim; j++)
      count = fscanf(file, "%f,", &(coords[i*spacedim+j]));
    count = fscanf(file, "-1\n");
  }
  count++;
}

//Write Output on file
void writeOutput(float* coords, int* classes, int spacedim, int numels)
{
  FILE *fp;
  fp = fopen(OUTPUT_FILE, "w");
  int i, j;
  for( i = 0; i < numels; i++)
  {
    for (j = 0; j < spacedim; j++)
      fprintf(fp, "%lf,", coords[i*spacedim+j]);
    
    fprintf(fp, "%d\n", classes[i]);
  }
  fclose(fp); 
}

//multidimensional euclidian distance (without sqrt)
__device__ float distance(float* const coords, float* const coords2, const int spacedim)
{
  float sum = 0;
  int i;
  for (i = 0; i < spacedim; i++)
  {
    float diff = coords[i] - coords2[i];
    sum += diff*diff;
  }  
  return sum;
}
